
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define dimx 8
#define dimy 8
#define dimz 4

typedef struct
{
    int Nx, Ny, Nz;
    double Lx, Ly, Lz;
    double c;
    double dx, dy, dz;
} Params;

static inline __host__ __device__ int get_index(const Params *P, int x, int y, int z)
{
    // rewrite to fit with threadid and stuff.
    return (z * P->Ny * P->Nx) + (y * P->Nx) + x;
}

// Laplacian at (i,j,k)
__device__ double laplacian(const Params *P, float *arr, int i, int j, int k)
{
    if (i <= 0 || i >= P->Nx - 1 ||
        j <= 0 || j >= P->Ny - 1 ||
        k <= 0 || k >= P->Nz - 1)
        return 0.0;
    
    int idx = get_index(P, i, j, k);
    double lap = 0.0;
    // Second-order central differences for 3D Laplacian
    // d2u/dx2
    lap += (arr[get_index(P, i + 1, j, k)] - 2 * arr[idx] + arr[get_index(P, i - 1, j, k)]) / (P->dx * P->dx);
    // + d2u/dy2
    lap += (arr[get_index(P, i, j + 1, k)] - 2 * arr[idx] + arr[get_index(P, i, j - 1, k)]) / (P->dy * P->dy);
    // + d2u/dz2
    lap += (arr[get_index(P, i, j, k + 1)] - 2 * arr[idx] + arr[get_index(P, i, j, k - 1)]) / (P->dz * P->dz);
    return lap;
}

// Initial condition: Spherical outgoing wave
void set_initial_conditions(const Params *P, float *u, float *v)
{
    double c = P->c;
    double eps = 1e-6; // avoid dividing by 0 in centre.
    // vectorise
    for (int k = 0; k < P->Nz; k++)
    {
        double z = k * P->dz - 0.5 * P->Lz;
        for (int j = 0; j < P->Ny; j++)
        {
            double y = j * P->dy - 0.5 * P->Ly;
            for (int i = 0; i < P->Nx; i++)
            {
                double x = i * P->dx - 0.5 * P->Lx;
                int idx = get_index(P, i, j, k);
                double r = sqrt(x * x + y * y + z * z);
                u[idx] = sin(10.0 * r) / (r + eps);
                v[idx] = -10.0 * c * cos(10.0 * r) / (r + eps); // du/dt at t=0
            }
        }
    }
}

// Write 3D data to file
void write_3d_data(const Params *P, float *arr, const char *filename)
{
    FILE *f = fopen(filename, "w");
    if (!f)
    {
        perror("Failed to open file for writing");
        return;
    }
    for (int k = 0; k < P->Nz; k++)
    {
        double z = k * P->dz - 0.5 * P->Lz;
        for (int j = 0; j < P->Ny; j++)
        {
            double y = j * P->dy - 0.5 * P->Ly;
            for (int i = 0; i < P->Nx; i++)
            {
                double x = i * P->dx - 0.5 * P->Lx;
                int idx = get_index(P, i, j, k);
                fprintf(f, "%f %f %f %f\n", x, y, z, arr[idx]);
            }
        }
        fprintf(f, "\n");
    }
    fclose(f);
}

static inline __device__ int get_coords(const Params *P, int *x_p, int *y_p, int *z_p)
{
    // Compute the offset in each dimension
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int z = blockDim.z * blockIdx.z + threadIdx.z;

    if (x >= P->Nx || y >= P->Ny || z >= P->Nz)
        return 0;

    *x_p = x;
    *y_p = y;
    *z_p = z;
    return 1;
}

__global__ void vec_add_uv(const Params *P, float *ku, float *kv, float *u, float *v)
{
    int x, y, z;
    printf("hello\n");

    if (get_coords(P, &x, &y, &z) == 0)
    {
        printf("out of bounds");
        return; // Thread out of bounds
    }
    int idx = get_index(P, x, y, z);
    printf("%i", idx);
    ku[idx] = v[idx];
    printf("inside gpu: ku[%i]: %f\n", idx, ku[idx]);
    kv[idx] = P->c * P->c * laplacian(P, u, x, y, z);
    printf("inside gpu: kv[%i]: %f\n", idx, kv[idx]);
}

__global__ void vec_add_rk(const Params *P, float *u, float *v, float *u_tmp, float *v_tmp, double dt, float *ku, float *kv)
{
    int x, y, z;
    get_coords(P, &x, &y, &z);
    int idx = get_index(P, x, y, z);
    u_tmp[idx] = u[idx] + dt * ku[idx];
    v_tmp[idx] = v[idx] + dt * kv[idx];
}

__global__ void vec_add_frk(const Params *P, float *u, float *v, double dt, float *ku1, float *kv1, float *ku2, float *kv2, float *ku3, float *kv3, float *ku4, float *kv4)
{
    int x, y, z;
    get_coords(P, &x, &y, &z);
    int idx = get_index(P, x, y, z);
    u[idx] += (dt / 6.0) * (ku1[idx] + 2 * ku2[idx] + 2 * ku3[idx] + ku4[idx]);
    v[idx] += (dt / 6.0) * (kv1[idx] + 2 * kv2[idx] + 2 * kv3[idx] + kv4[idx]);
}

// RK4 step for the wave equation
void rk4_wave_parallelised(const Params *P, float *u, float *v, float *u_tmp, float *v_tmp, double dt)
{
    dim3 block(dimx, dimy, dimz);
    dim3 grid((P->Nx + dimx - 1) / dimx, (P->Ny + dimy - 1) / dimy, (P->Nz + dimz - 1) / dimz);
    int N = P->Nx * P->Ny * P->Nz;
    int size = N * sizeof(float);
    float *ku1;
    hipMalloc(&ku1, size);
    float *kv1;
    hipMalloc(&kv1, size);
    float *ku2;
    hipMalloc(&ku2, size);
    float *kv2;
    hipMalloc(&kv2, size);
    float *ku3;
    hipMalloc(&ku3, size);
    float *kv3;
    hipMalloc(&kv3, size);
    float *ku4;
    hipMalloc(&ku4, size);
    float *kv4;
    hipMalloc(&kv4, size);

    // k1
    vec_add_uv<<<grid, block>>>(P, ku1, kv1, u, v);

    // k2
    vec_add_rk<<<grid, block>>>(P, u, v, u_tmp, v_tmp, 0.5 * dt, ku1, kv1);
    vec_add_uv<<<grid, block>>>(P, ku2, kv2, u_tmp, v_tmp);

    // k3
    vec_add_rk<<<grid, block>>>(P, u, v, u_tmp, v_tmp, 0.5 * dt, ku2, kv2);
    vec_add_uv<<<grid, block>>>(P, ku3, kv3, u_tmp, v_tmp);

    // k4
    vec_add_rk<<<grid, block>>>(P, u, v, u_tmp, v_tmp, dt, ku3, kv3);
    vec_add_uv<<<grid, block>>>(P, ku4, kv4, u_tmp, v_tmp);

    // Update u and v
    vec_add_frk<<<grid, block>>>(P, u, v, dt, ku1, kv1, ku2, kv2, ku3, kv3, ku4, kv4);

    hipFree(ku1);
    hipFree(kv1);
    hipFree(ku2);
    hipFree(kv2);
    hipFree(ku3);
    hipFree(kv3);
    hipFree(ku4);
    hipFree(kv4);
}

int main()
{
    Params P = {
        .Nx = 30,
        .Ny = 30,
        .Nz = 30,
        .Lx = 2.0,
        .Ly = 2.0,
        .Lz = 2.0,
        .c = 1.0};
    P.dx = P.Lx / (P.Nx - 1);
    P.dy = P.Ly / (P.Ny - 1);
    P.dz = P.Lz / (P.Nz - 1);

    int steps = 20;
    double dt = 1.0 / steps; // must be changed

    // stability check
    double cmax = sqrt(2);
    double h = sqrt(1 / (P.dx * P.dx) + 1 / (P.dy * P.dy) + 1 / (P.dz * P.dz));

    if ((P.c * dt) / h > cmax)
    {
        printf("Unstable simulation. Exiting. \n");
        return 0;
    }

    Params *d_P;
    hipMalloc(&d_P, sizeof(Params));
    hipMemcpy(d_P, &P, sizeof(Params), hipMemcpyHostToDevice);

    int N = P.Nx * P.Ny * P.Nz;
    int size = N * sizeof(float);
    float *h_u = (float *)calloc(N, sizeof(float));
    float *h_v = (float *)calloc(N, sizeof(float));

    set_initial_conditions(&P, h_u, h_v);

    float *d_u;
    hipMalloc(&d_u, size);
    float *d_v;
    hipMalloc(&d_v, size);

    hipMemcpy(d_u, h_u, size, hipMemcpyHostToDevice);
    hipMemcpy(d_v, h_v, size, hipMemcpyHostToDevice);

    float *d_u_tmp;
    hipMalloc(&d_u_tmp, size);
    float *d_v_tmp;
    hipMalloc(&d_v_tmp, size);

    system("mkdir -p timestep");

    for (int s = 0; s < steps; s++)
    {
        // save results. Done before in order to also save init
        char fname[64];
        snprintf(fname, sizeof(fname), "timestep/wave3d_%03d.txt", s);
        write_3d_data(&P, h_u, fname);

        // note, last timestep isn't written to file lol.
        rk4_wave_parallelised(d_P, d_u, d_v, d_u_tmp, d_v_tmp, dt);
        hipDeviceSynchronize();

        hipMemcpy(h_u, d_u, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_v, d_v, size, hipMemcpyDeviceToHost);
    }

    // save final timestep
    char fname[64];
    snprintf(fname, sizeof(fname), "timestep/wave3d_%03d.txt", steps);
    write_3d_data(&P, h_u, fname);

    // free host memory
    free(h_u);
    free(h_v);

    // free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_u_tmp);
    hipFree(d_v_tmp);

    return 0;
}